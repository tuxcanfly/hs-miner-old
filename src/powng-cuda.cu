#include <assert.h>
#include <unistd.h>
#include <stdbool.h>
#include <stdint.h>
#include "common.h"

int32_t
hs_cuda_run(
  hs_options_t *options,
  uint8_t *solution,
  bool *match
) {
#ifdef HS_HAS_CUDA
  uint8_t header[MAX_HEADER_SIZE];
  size_t header_len = options->header_len;
  uint32_t nonce = options->nonce;
  uint32_t range = 1;
  uint32_t device = options->device;
  uint8_t hash[32];
  uint8_t chash[32];

  memset(hash, 0xff, 32);

  if (header_len < MIN_HEADER_SIZE || header_len > MAX_HEADER_SIZE)
    return HS_EBADARGS;

  memcpy(header, options->header, header_len);

  int32_t device_count = 0;
  hipGetDeviceCount(&device_count);

  if (device_count < 0 || device >= device_count)
    return HS_ENODEVICE;

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);

  if (options->range)
    range = options->range;

  hipSetDevice(device);

  *match = false;

  for (uint32_t r = 0; r < range; r++) {
    if (!options->running)
      break;

    int32_t rc = verify(sol, &ctx.trimmer->sip_keys);

    if (rc == POW_OK) {
      return HS_SUCCESS;
    }

    if (memcmp(chash, options->target, 32) <= 0) {
      *match = true;
      return HS_SUCCESS;
    }
  }

  return HS_ENOSOLUTION;
#else
  return HS_ENOSUPPORT;
#endif
}
