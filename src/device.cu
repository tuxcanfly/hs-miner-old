#include <assert.h>
#include <stdbool.h>
#include "common.h"

uint32_t
hs_cuda_device_count(void) {
  int32_t device_count = 0;

  hipGetDeviceCount(&device_count);

  if (device_count < 0)
    return 0;

  return device_count;
}

bool
hs_cuda_device_info(uint32_t device, hs_device_info_t *info) {
  assert(info);

  int32_t device_count = 0;

  hipGetDeviceCount(&device_count);

  if (device_count < 0 || device >= device_count)
    return false;

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);

  if (strlen(prop.name) > 512)
    return false;

  strcpy(info->name, prop.name);

  info->memory = prop.totalGlobalMem;
  info->bits = prop.memoryBusWidth;
  info->clock_rate = prop.memoryClockRate;

  return true;
}
